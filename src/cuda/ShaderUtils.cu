#include "hip/hip_runtime.h"
#include "ShaderUtils.h"
#include "CudaData.h"
namespace sharc
{
    __device__ void get_cam_normals(const int& i, const int& j, float* n1, float* n2, float* n3)
    {
        float screen_xyz;
        float i_portion = 0.5 - ((float)i)/(i_bound-1);
        float j_portion = ((float)j)/(j_bound-1) - 0.5;

        //"X" (rot_sin, -rot_cos, 0)
        //"Y" (-rot_cos*elev_sin, -rot_sin*elev_sin, elev_sin)
        screen_xyz = cam_x + zoom*cam_n1 + j_portion*rot_sin*zoom + i_portion*(-rot_cos*elev_sin);
        *n1 = screen_xyz-cam_x;

        screen_xyz = cam_y + zoom*cam_n2 + j_portion*(-rot_cos)*zoom + i_portion*(-rot_sin*elev_sin);
        *n2 = screen_xyz-cam_y;

        screen_xyz = cam_z + zoom*cam_n3 + i_portion*elev_cos;
        *n3 = screen_xyz-cam_z;

        screen_xyz = sqrt((*n1)*(*n1) + (*n2)*(*n2) + (*n3)*(*n3));
        *n1 /= screen_xyz;
        *n2 /= screen_xyz;
        *n3 /= screen_xyz;
    }
}
