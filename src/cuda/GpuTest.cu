#include "hip/hip_runtime.h"
#include <iostream>
#include "Config.h"
#include "GpuTest.h"
#include "CudaErrorLogging.h"
#include "sharcCuda.h"
#include "PngWriter.h"
namespace sharc
{
    #define FRAC_XMIN -1.25086
	#define FRAC_XMAX -1.25046
	#define FRAC_YMIN  0.02002
	#define FRAC_YMAX  0.02022
	#define FRAC_ITS 1600
	#define FRAC_RATIO 0.4
    __global__ void K_gpu_test(int* imdata, const int wid, const int hei)
    {
        int row = blockIdx.y*blockDim.y + threadIdx.y;
		int col = blockIdx.x*blockDim.x + threadIdx.x;
        if (row < hei && col < wid)
		{
			double x0 = FRAC_XMIN - (FRAC_XMIN-FRAC_XMAX)*(double)col/(double)wid;
			double y0 = FRAC_YMIN - (FRAC_YMIN-FRAC_YMAX)*(double)row/(double)hei;
			double x = x0;
			double y = y0;
			double temp;
			int ct = 0;
			bool diverged = false;
			while (ct++ < FRAC_ITS && !diverged)
			{
				temp = x*x - y*y + x0;
				y = 2*x*y + y0;
				x = temp;
				if (x*x + y*y > 4)
				{
					double ratio = (double)ct/FRAC_ITS;
					ratio = exp(0.7*log(ratio));
					double ratio_loc = 0;
					if (ratio > FRAC_RATIO)
					{
						ratio_loc = (ratio - FRAC_RATIO) / (1-FRAC_RATIO);
						ratio_loc = sqrt(ratio_loc);
                        *(imdata + row*wid + col) = rgb2int(1, sqrt(ratio_loc), ratio_loc*ratio_loc);
					}
					else
					{
						ratio_loc = ratio/FRAC_RATIO;
						ratio_loc = ratio_loc*ratio_loc;
                        *(imdata + row*wid + col) = rgb2int(ratio_loc, 0.4*(ratio_loc)*(1 - ratio_loc), 0.2*(1 - ratio_loc));
					}
					diverged = true;
				}
			}
			if (!diverged)
			{
				*(imdata + row*wid + col) = rgb2int(1, 1, 1);
			}
		}
    }

    void gpu_test(void)
    {
        int wid = SHARC_TEST_WID;
        int hei = SHARC_TEST_HEI;
        int* d_image_data;
        int* imdata = (int*)malloc(wid*hei*sizeof(int));
        CU_CHK(hipMalloc(&d_image_data, wid*hei*sizeof(int)));

        int num_blocks_w = (wid + (BLOCK_SIZE-1))/BLOCK_SIZE;
        int num_blocks_h = (hei + (BLOCK_SIZE-1))/BLOCK_SIZE;

        dim3 grid(num_blocks_w, num_blocks_h);
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);

        K_gpu_test<<<grid, block>>>(d_image_data, wid, hei);
        CU_CHK(hipPeekAtLastError());
        CU_CHK(hipDeviceSynchronize());
        CU_CHK(hipMemcpy(imdata, d_image_data, wid*hei*sizeof(int), hipMemcpyDeviceToHost));
        CU_CHK(hipFree(d_image_data));
        PngWriter p;
        p.write_png("fractaltest.png", wid, hei, imdata);
        free(imdata);
    }
}
