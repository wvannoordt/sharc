#include "hip/hip_runtime.h"
#include <iostream>
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "sharcCuda.h"
#include "Shader_TestShader.h"
#include "CuGLInterop.h"
namespace sharc
{
    __global__ void K_rdr_test_shader()
    {
        //How to best make these into macros??
        int i = blockIdx.y*blockDim.y + threadIdx.y;
		int j = blockIdx.x*blockDim.x + threadIdx.x;
        float r,g,b;
        r = (float)((blockIdx.y + blockIdx.x)%2);
        g = 0.0;
        b = (float)((blockIdx.y/2 + blockIdx.x/2)%2);
        if (i < layers.Ni && j < layers.Nj)
		{
            *(layers.imdata + i*layers.Nj+j) = rgb2int(r, g, b);
		}
    }

    void shader_testshader(void)
    {
        K_rdr_test_shader<<<grid_conf, block_conf>>>();
        CU_CHK(hipPeekAtLastError());
        CU_CHK(hipDeviceSynchronize());
    }
}
