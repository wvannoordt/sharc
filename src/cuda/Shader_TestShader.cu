#include "hip/hip_runtime.h"
#include <iostream>
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "sharcCuda.h"
namespace sharc
{
    __global__ void K_rdr_test_shader()
    {
        //How to best make these into macros??
        int i = blockIdx.y*blockDim.y + threadIdx.y;
		int j = blockIdx.x*blockDim.x + threadIdx.x;
        float r;
        r = (float)((blockIdx.y + blockIdx.x)%2);
        if (i < i_bound && j < j_bound)
		{
            *(imdata + i*j_bound+j) = rgb2int(r, r, r);
		}
    }

    void shader_testshader(void)
    {
        K_rdr_test_shader<<<grid_conf, block_conf>>>();
        CU_CHK(hipDeviceSynchronize());
    }
}
