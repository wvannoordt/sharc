#include "hip/hip_runtime.h"
#include <iostream>
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "sharcCuda.h"
#include "Shader_TestShader.h"
#include "CuGLInterop.h"
namespace sharc
{
    __global__ void K_rdr_TestShader()
    {
        //How to best make these into macros??
        int i = blockIdx.y*blockDim.y + threadIdx.y;
		int j = blockIdx.x*blockDim.x + threadIdx.x;
        float r,g,b;
        r = (float)((blockIdx.y + blockIdx.x)%2);
        g = 0.0;
        b = (float)((blockIdx.y/2 + blockIdx.x/2)%2);
        if (i < layers.Ni && j < layers.Nj)
		{
            *(layers.imdata + i*layers.Nj+j) = rgb2int(r, g, b);
		}
    }

    void Shader_TestShader(void)
    {
        K_rdr_TestShader<<<gridConf, blockConf>>>();
        CU_CHK(hipPeekAtLastError());
        CU_CHK(hipDeviceSynchronize());
    }
}
