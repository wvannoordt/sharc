#include "hip/hip_runtime.h"
#include <iostream>
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "sharcCuda.h"
#include "Shader_Environ.h"
#include "ShaderUtils.h"
namespace sharc
{
    __global__ void K_rdr_EnvironShader()
    {
        //How to best make these into macros??
        int i = blockIdx.y*blockDim.y + threadIdx.y;
		int j = blockIdx.x*blockDim.x + threadIdx.x;
        vec3 norm;
        if (i < layers.Ni && j < layers.Nj)
		{
            GetCamNormals(i, j, norm);
            *(layers.imdata + i*layers.Nj+j) = (norm[2]>0) ? settings.sky_color : settings.floor_color;
            if (settings.rdr_floor && norm[2] < 0)
            {
                int floorx, floory;
                float trav = (settings.floor_height - settings.cam_z)/(norm[2]);
                floorx = (int)floor((0.1*(trav*norm[0]+settings.cam_x)));
                floory = (int)floor((0.1*(trav*norm[1]+settings.cam_y)));
                floorx += floory;
                if ((floorx%2==0)==floorx<0) *(layers.imdata + i*layers.Nj+j) = 0xff341989;
                if (abs((0.1*(trav*norm[1]+settings.cam_y))) < 1e-2 || abs((0.1*(trav*norm[0]+settings.cam_x))) < 1e-2) *(layers.imdata + i*layers.Nj+j) = 0;
            }
		}
    }

    void Shader_Environ(void)
    {
        K_rdr_EnvironShader<<<gridConf, blockConf>>>();
        CU_CHK(hipPeekAtLastError());
        CU_CHK(hipDeviceSynchronize());
    }
}
