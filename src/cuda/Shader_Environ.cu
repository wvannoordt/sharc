#include "hip/hip_runtime.h"
#include <iostream>
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "sharcCuda.h"
#include "Shader_Environ.h"
#include "ShaderUtils.h"
namespace sharc
{
    __global__ void K_rdr_environ_shader()
    {
        //How to best make these into macros??
        int i = blockIdx.y*blockDim.y + threadIdx.y;
		int j = blockIdx.x*blockDim.x + threadIdx.x;
        vec3 norm;
        if (i < i_bound && j < j_bound)
		{
            get_cam_normals(i, j, norm);
            *(imdata + i*j_bound+j) = (norm[2]>0) ? skybackground : floorbackground;
            if (rdr_floor && norm[2] < 0)
            {
                int floorx, floory;
                float trav = (floor_height - cam_z)/(norm[2]);
                floorx = (int)floor((0.1*(trav*norm[0]+cam_x)));
                floory = (int)floor((0.1*(trav*norm[1]+cam_y)));
                floorx += floory;
                if ((floorx%2==0)==floorx<0) *(imdata + i*j_bound+j) = 0x00341989;
                if (abs((0.1*(trav*norm[1]+cam_y))) < 1e-2 || abs((0.1*(trav*norm[0]+cam_x))) < 1e-2)*(imdata + i*j_bound+j) = 0;
            }
		}
    }

    void shader_environ(void)
    {
        K_rdr_environ_shader<<<grid_conf, block_conf>>>();
        CU_CHK(hipPeekAtLastError());
        CU_CHK(hipDeviceSynchronize());
    }
}
