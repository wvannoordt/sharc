#include "hip/hip_runtime.h"
#include <iostream>
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "sharcCuda.h"
#include "Shader_Environ.h"
#include "ShaderUtils.h"
namespace sharc
{
    __global__ void K_rdr_environ_shader()
    {
        //How to best make these into macros??
        int i = blockIdx.y*blockDim.y + threadIdx.y;
		int j = blockIdx.x*blockDim.x + threadIdx.x;
        float n1, n2, n3;
        if (i < i_bound && j < j_bound)
		{
            get_cam_normals(i, j, &n1, &n2, &n3);
            *(imdata + i*j_bound+j) = (n3>0) ? skybackground : floorbackground;
		}
    }

    void shader_environ(void)
    {
        K_rdr_environ_shader<<<grid_conf, block_conf>>>();
        CU_CHK(hipPeekAtLastError());
        CU_CHK(hipDeviceSynchronize());
    }
}
