#include "sharcCuda.h"
#include "sharc.h"
#include "Typedef.h"
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "sharcShaders.h"

namespace sharc
{
    void cuda_allocate_frame_buffers(void)
    {
        CU_CHK(hipMalloc(&H_imdata, userSettings.width * userSettings.height * sizeof(int)));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(imdata), &(H_imdata), sizeof(int*), 0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(i_bound), &(userSettings.height), sizeof(int), 0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(j_bound), &(userSettings.width),  sizeof(int), 0));
        int num_blocks_w = (userSettings.width  + (BLOCK_SIZE-1))/BLOCK_SIZE;
        int num_blocks_h = (userSettings.height + (BLOCK_SIZE-1))/BLOCK_SIZE;
        grid_conf =  dim3(num_blocks_w, num_blocks_h);
        block_conf = dim3(BLOCK_SIZE, BLOCK_SIZE);
    }

    void cuda_compute_shader(const int shader_id)
    {
        switch (shader_id)
        {
            case SHARC_SHDR_TEST:
            {
                shader_testshader();
                break;
            }
        }
    }

    void offload_image_data(int* host_endpoint)
    {
        CU_CHK(hipMemcpy(host_endpoint, H_imdata, userSettings.width * userSettings.height * sizeof(int), hipMemcpyDeviceToHost));
    }

    void cuda_finalize(void)
    {
        CU_CHK(hipFree(H_imdata));
    }
}
