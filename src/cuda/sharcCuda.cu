#include "sharcCuda.h"
#include "sharc.h"
#include "Typedef.h"
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "sharcShaders.h"
#include "Shader_TestShader.h"
#include "Shader_Environ.h"

namespace sharc
{
    void CudaAllocateFrameBuffers(void)
    {
        AllocateFrameBuffers(userSettings.width, userSettings.height);
    }

    void CudaSetRenderState(void)
    {
        SetRenderState(&userSettings);
    }

    void CudaComputeShader(const int shader_id)
    {
        switch (shader_id)
        {
            case SHARC_SHDR_TEST:
            {
                Shader_TestShader();
                break;
            }
            case SHARC_SHDR_ENVIRON:
            {
                Shader_Environ();
                break;
            }
        }
    }

    void OffloadImageData(int* host_endpoint)
    {
        CU_CHK(hipMemcpy(host_endpoint, shaderLayers.imdata, userSettings.width * userSettings.height * sizeof(int), hipMemcpyDeviceToHost));
    }

    void CudaFinalize(void)
    {
        FreeFrameBuffers();
    }
}
