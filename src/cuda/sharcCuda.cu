#include "sharcCuda.h"
#include "sharc.h"
#include "Typedef.h"
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "sharcShaders.h"
#include "Shader_TestShader.h"
#include "Shader_Environ.h"

namespace sharc
{
    void cuda_allocate_frame_buffers(void)
    {
        allocate_frame_bufs(userSettings.width, userSettings.height);
    }

    void cuda_set_render_state(void)
    {
        set_render_state(&userSettings);
    }

    void cuda_compute_shader(const int shader_id)
    {
        switch (shader_id)
        {
            case SHARC_SHDR_TEST:
            {
                shader_testshader();
                break;
            }
            case SHARC_SHDR_ENVIRON:
            {
                shader_environ();
                break;
            }
        }
    }

    void offload_image_data(int* host_endpoint)
    {
        CU_CHK(hipMemcpy(host_endpoint, shaderLayers.imdata, userSettings.width * userSettings.height * sizeof(int), hipMemcpyDeviceToHost));
    }

    void cuda_finalize(void)
    {
        free_frame_bufs();
    }
}
