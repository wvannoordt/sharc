#include "hip/hip_runtime.h"
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "Typedef.h"
namespace sharc
{
    //camera position
    __device__ float cam_x, cam_y, cam_z, elev_cos, elev_sin, rot_cos, rot_sin, zoom, cam_n1, cam_n2, cam_n3, screen_aspect, floor_height;
    __device__ bool rdr_floor;

    __device__ int skybackground, floorbackground;

    dim3 grid_conf, block_conf;
    int* H_imdata;

    //image data
    __device__ int* imdata;
    __device__ int i_bound, j_bound;

    ////////shader buffer layers
    //incident coordinates
    g_real *H_x_inc, *H_y_inc, *H_z_inc;
    __device__ g_real *x_inc, *y_inc, *z_inc;

    //incident normals
    float *H_n1_inc, *H_n2_inc, *H_n3_inc;
    __device__ float *n1_inc, *n2_inc, *n3_inc;

    //buffer indices
    uint16* H_ob_id;
    __device__ uint16* ob_id;

    void allocate_frame_bufs(int wid, int hei)
    {
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(j_bound), &(userSettings.width),  sizeof(int), 0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(i_bound), &(userSettings.height), sizeof(int), 0));

        CU_CHK(hipMalloc((void**)(&H_imdata), wid * hei * sizeof(int)));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(imdata), &(H_imdata), sizeof(int*), 0));

        CU_CHK(hipMalloc((void**)(&H_x_inc), wid * hei * sizeof(g_real)));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(x_inc), &(H_x_inc), sizeof(g_real*), 0));

        CU_CHK(hipMalloc((void**)(&H_y_inc), wid * hei * sizeof(g_real)));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(y_inc), &(H_y_inc), sizeof(g_real*), 0));

        CU_CHK(hipMalloc((void**)(&H_z_inc), wid * hei * sizeof(g_real)));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(z_inc), &(H_z_inc), sizeof(g_real*), 0));

        CU_CHK(hipMalloc((void**)(&H_ob_id), wid * hei * sizeof(uint16)));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(ob_id), &(H_ob_id), sizeof(uint16*), 0));

        int num_blocks_w = (userSettings.width  + (BLOCK_SIZE-1))/BLOCK_SIZE;
        int num_blocks_h = (userSettings.height + (BLOCK_SIZE-1))/BLOCK_SIZE;
        grid_conf =  dim3(num_blocks_w, num_blocks_h);
        block_conf = dim3(BLOCK_SIZE, BLOCK_SIZE);
    }

    void free_frame_bufs(void)
    {
        CU_CHK(hipFree(H_imdata));
        CU_CHK(hipFree(H_x_inc));
        CU_CHK(hipFree(H_y_inc));
        CU_CHK(hipFree(H_z_inc));
        CU_CHK(hipFree(H_ob_id));
    }

    void set_render_state(SharcSettings* settings)
    {
        float s_elev = sin(settings->cam_elev);
        float c_elev = cos(settings->cam_elev);
        float s_rot  = sin(settings->cam_rot);
        float c_rot  = cos(settings->cam_rot);
        float H_cam_n1 = c_elev*c_rot;
        float H_cam_n2 = c_elev*s_rot;
        float H_cam_n3 = s_elev;
        float scr_asp = (float)((double)userSettings.height / (double)userSettings.width);

        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(screen_aspect), &(scr_asp),  sizeof(float), 0));

        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(cam_x), &(settings->cam_x),  sizeof(float), 0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(cam_y), &(settings->cam_y),  sizeof(float), 0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(cam_z), &(settings->cam_z),  sizeof(float), 0));

        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(rdr_floor),    &(settings->rdr_floor),  sizeof(bool), 0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(floor_height), &(settings->floor_height),  sizeof(float), 0));

        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(cam_n1), &(H_cam_n1),  sizeof(float), 0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(cam_n2), &(H_cam_n2),  sizeof(float), 0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(cam_n3), &(H_cam_n3),  sizeof(float), 0));

        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(elev_sin), &(s_elev),  sizeof(float), 0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(elev_cos), &(c_elev),  sizeof(float), 0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(rot_sin),  &(s_rot),   sizeof(float), 0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(rot_cos),  &(c_rot),   sizeof(float), 0));

        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(skybackground),    &(settings->sky_color),     sizeof(int),   0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(floorbackground),  &(settings->floor_color),   sizeof(int),   0));
        CU_CHK(hipMemcpyToSymbol(HIP_SYMBOL(zoom),             &(settings->zoom_aspect),   sizeof(float), 0));
    }
}
