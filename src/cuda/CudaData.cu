#include "hip/hip_runtime.h"
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "Typedef.h"
namespace sharc
{
    __device__ SharcSettings settings;
    __device__ SharcShaderLayers layers;dim3 grid_conf, block_conf;

    __global__ void K_settings(SharcSettings s)
    {
        settings = s;
    }

    __global__ void K_layers(SharcShaderLayers s)
    {
        layers = s;
    }

    void allocate_frame_bufs(int wid, int hei)
    {
        CU_CHK(hipMalloc((void**)(&(shaderLayers.imdata)),        wid * hei * sizeof(uchar4)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_x[0])), wid * hei * sizeof(g_real)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_x[1])), wid * hei * sizeof(g_real)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_x[2])), wid * hei * sizeof(g_real)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_n[0])), wid * hei * sizeof(float)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_n[1])), wid * hei * sizeof(float)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_n[2])), wid * hei * sizeof(float)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.object_id)),     wid * hei * sizeof(uint16)));
        shaderLayers.Nj = wid;
        shaderLayers.Ni = hei;

        int num_blocks_w = (userSettings.width  + (BLOCK_SIZE-1))/BLOCK_SIZE;
        int num_blocks_h = (userSettings.height + (BLOCK_SIZE-1))/BLOCK_SIZE;
        grid_conf =  dim3(num_blocks_w, num_blocks_h);
        block_conf = dim3(BLOCK_SIZE, BLOCK_SIZE);
        K_layers<<<1,1>>>(shaderLayers);
    }

    void free_frame_bufs(void)
    {
        CU_CHK(hipFree(shaderLayers.imdata));
        CU_CHK(hipFree(shaderLayers.incident_x[0]));
        CU_CHK(hipFree(shaderLayers.incident_x[1]));
        CU_CHK(hipFree(shaderLayers.incident_x[2]));
        CU_CHK(hipFree(shaderLayers.incident_n[0]));
        CU_CHK(hipFree(shaderLayers.incident_n[1]));
        CU_CHK(hipFree(shaderLayers.incident_n[2]));
        CU_CHK(hipFree(shaderLayers.object_id));
    }

    void set_render_state(SharcSettings* settings_in)
    {
        settings_in->elev_sin = sin(settings_in->cam_elev);
        settings_in->elev_cos = cos(settings_in->cam_elev);
        settings_in->rot_sin  = sin(settings_in->cam_rot);
        settings_in->rot_cos  = cos(settings_in->cam_rot);
        settings_in->cam_n[0] = (settings_in->elev_cos)*(settings_in->rot_cos);
        settings_in->cam_n[1] = (settings_in->elev_cos)*(settings_in->rot_sin);
        settings_in->cam_n[2] = (settings_in->elev_sin);
        K_settings<<<1,1>>>(*settings_in);
    }
}
