#include "hip/hip_runtime.h"
#include "CudaData.h"
#include "CudaErrorLogging.h"
#include "Typedef.h"
#include "sharc.h"
namespace sharc
{
    __device__ SharcSettings settings;
    __device__ SharcShaderLayers layers;
    dim3 gridConf, blockConf;

    __global__ void K_settings(SharcSettings s)
    {
        settings = s;
    }

    __global__ void K_layers(SharcShaderLayers s)
    {
        layers = s;
    }

    void AllocateFrameBuffers(int wid, int hei)
    {
        if (!interactiveMode) CU_CHK(hipMalloc((void**)(&(shaderLayers.imdata)), wid * hei * sizeof(int)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_x[0])), wid * hei * sizeof(g_real)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_x[1])), wid * hei * sizeof(g_real)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_x[2])), wid * hei * sizeof(g_real)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_n[0])), wid * hei * sizeof(float)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_n[1])), wid * hei * sizeof(float)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.incident_n[2])), wid * hei * sizeof(float)));
        CU_CHK(hipMalloc((void**)(&(shaderLayers.object_id)),     wid * hei * sizeof(uint16)));
        shaderLayers.Nj = wid;
        shaderLayers.Ni = hei;

        int num_blocks_w = (userSettings.width  + (BLOCK_SIZE-1))/BLOCK_SIZE;
        int num_blocks_h = (userSettings.height + (BLOCK_SIZE-1))/BLOCK_SIZE;
        gridConf =  dim3(num_blocks_w, num_blocks_h);
        blockConf = dim3(BLOCK_SIZE, BLOCK_SIZE);
        K_layers<<<1,1>>>(shaderLayers);
    }

    void FreeFrameBuffers(void)
    {
        CU_CHK(hipFree(shaderLayers.imdata));
        CU_CHK(hipFree(shaderLayers.incident_x[0]));
        CU_CHK(hipFree(shaderLayers.incident_x[1]));
        CU_CHK(hipFree(shaderLayers.incident_x[2]));
        CU_CHK(hipFree(shaderLayers.incident_n[0]));
        CU_CHK(hipFree(shaderLayers.incident_n[1]));
        CU_CHK(hipFree(shaderLayers.incident_n[2]));
        CU_CHK(hipFree(shaderLayers.object_id));
    }

    void SetRenderState(SharcSettings* settings_in)
    {
        settings_in->elev_sin = sin(settings_in->cam_elev);
        settings_in->elev_cos = cos(settings_in->cam_elev);
        settings_in->rot_sin  = sin(settings_in->cam_rot);
        settings_in->rot_cos  = cos(settings_in->cam_rot);
        settings_in->cam_n[0] = (settings_in->elev_cos)*(settings_in->rot_cos);
        settings_in->cam_n[1] = (settings_in->elev_cos)*(settings_in->rot_sin);
        settings_in->cam_n[2] = (settings_in->elev_sin);
        K_settings<<<1,1>>>(*settings_in);
    }

    __global__ void K_OGLFrameBuffer(int* buf)
    {
        layers.imdata = buf;
    }

    void OverrideOGLFrameBuffer(int* newBuf)
    {
        K_OGLFrameBuffer<<<1,1>>>(newBuf);
    }
}
